// To compile this sample code:
//
// nvcc gds_helloworld.cxx -o gds_helloworld -lcufile
//
// Set the environment variable TESTFILE
// to specify the name of the file on a GDS enabled filesystem
//
// Ex:   TESTFILE=/mnt/gds/gds_test ./gds_helloworld
//
//
#include <fcntl.h>
#include <errno.h>
#include <unistd.h>

#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <random>

#include <iostream>

#include <hip/hip_runtime.h>
#include <nvcufile.h>

#include "hip/hip_runtime_api.h"

using namespace std;

int main(void) {
    hipError_t cuda_result;
    ssize_t ret;
    CUfileError_t status;

    off_t file_offset = 0x0;
    off_t devPtr_offset = 0x0;
    ssize_t IO_size = 4096ULL;
    size_t buff_size = IO_size + 0x0;

    int d = 64;      // dimension
    int nb = 100000; // database size
    int nq = 10000;  // nb of queries
    float* host_data = new float[d * nb];

    std::mt19937 rng;
    std::uniform_real_distribution<> distrib;

    printf("Filling memory.\n");
    float* xb = new float[d * nb];
    float* xq = new float[d * nq];

    for (int i = 0; i < nb; i++) {
        for (int j = 0; j < d; j++)
            xb[d * i + j] = distrib(rng);
        xb[d * i] += i / 1000.;
    }

    for (int i = 0; i < nq; i++) {
        for (int j = 0; j < d; j++)
            xq[d * i + j] = distrib(rng);
        xq[d * i] += i / 1000.;
    }

    printf("Allocating CUDA buffer\n");
    float *dev_xb, *dev_xq;
    checkCudaErrors(hipMalloc(&dev_xb, d * nb * sizeof(float)));
    checkCudaErrors(hipMalloc(&dev_xq, d * nq * sizeof(float)));

    checkCudaErrors(hipMemcpy(dev_xb, xb, d * nb * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_xq, xq, d * nq * sizeof(float), hipMemcpyHostToDevice));

    int base_fd = open("/mnt/nvmetest/faiss_data_trained.c", O_CREAT | O_RDWR | O_DIRECT, 0644);
    if(base_fd < 0) {
        printf("file open errno %d\n", errno);
        return -1;
    }
    int query_fd = open("/mnt/nvmetest/faiss_data_query.c", O_CREAT | O_RDWR | O_DIRECT, 0644);
    if(query_fd < 0) {
        printf("file open errno %d\n", errno);
        return -1;
    }

    printf("Opening cuFileDriver.\n");
    status = cuFileDriverOpen();
    if (status.err != CU_FILE_SUCCESS) {
        printf(" cuFile driver failed to open \n");
        goto cufile_open_failed;
    }

    printf("Registering cuFile handle.\n");
    CUfileDescr_t cf_descr;
    CUfileHandle_t cf_handle;
    memset((void *)&cf_descr, 0, sizeof(CUfileDescr_t));
    cf_descr.handle.fd = base_fd;
    cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
    status = cuFileHandleRegister(&cf_handle, &cf_descr);
    if (status.err != CU_FILE_SUCCESS) {
        std::cerr << "cuFileHandleRegister base_fd " << base_fd << " status " << status.err << std::endl;
        goto handle_register_failed;
    }

    printf(" Registering Buffer of %lu bytes and %lu bytes.\n",
        d * nb * sizeof(float), d * nq * sizeof(float));
    status = cuFileBufRegister(dev_xb, d * nb * sizeof(float), 0);
    if (status.err != CU_FILE_SUCCESS) {
        printf("buffer registration failed %d\n", status.err);
        goto register_failed;
    }
    status = cuFileBufRegister(dev_xq, d * nq * sizeof(float), 0);
    if (status.err != CU_FILE_SUCCESS) {
        printf("buffer registration failed %d\n", status.err);
        goto register_failed;
        return -1;
    }

    // perform write operation directly from GPU mem to file
    printf("Writing buffer to file.\n");
    ret = cuFileWrite(cf_handle, dev_xb, d * nb * sizeof(float), 0, 0);
    if (ret < 0 || ret != d * nb * sizeof(float)) {
        printf("cuFileWrite failed %zu\n", ret);
        goto write_fail;
    }
    
    // print file data
    checkCudaErrors(hipMemcpy(host_data, dev_xb, d * nb * sizeof(float), hipMemcpyDeviceToHost));
    for (size_t i = 0; i < d * nb; i++) {
        if (i / d >= 100) {
            printf("...\n");
            break;
        }
        printf("%.2f ", host_data[i]);
        if ((i + 1) % d == 0) {
            printf("\n");
        }
    }

    // release the GPU memory pinning
    printf("Releasing cuFile buffer.\n");
    status = cuFileBufDeregister(dev_xb);
    if (status.err != CU_FILE_SUCCESS) {
        std::cerr << "buffer deregister failed" << std::endl;
        hipFree(dev_xb);
        cuFileHandleDeregister(cf_handle);
        close(base_fd);
        return -1;
    }
    status = cuFileBufDeregister(dev_xq);
    if (status.err != CU_FILE_SUCCESS) {
        std::cerr << "buffer deregister failed" << std::endl;
        hipFree(dev_xb);
        cuFileHandleDeregister(cf_handle);
        close(base_fd);
        return -1;
    }

    printf("Freeing CUDA buffer.\n");
    checkCudaErrors(hipFree(dev_xb));
    checkCudaErrors(hipFree(dev_xq));

        // deregister the handle from cuFile
        cout << "Releasing file handle. " << std::endl;
        (void) cuFileHandleDeregister(cf_handle);
        close(base_fd);

        // release all cuFile resources
        cout << "Closing File Driver." << std::endl;
        (void) cuFileDriverClose();
        cout << std::endl;

    return 0;

write_fail:
register_failed:
    cuFileHandleDeregister(cf_handle);
    hipFree(dev_xb);
    hipFree(dev_xq);
handle_register_failed:
    (void) cuFileDriverClose();
cufile_open_failed:
    close(base_fd);
    close(query_fd);

    return -1;
}
